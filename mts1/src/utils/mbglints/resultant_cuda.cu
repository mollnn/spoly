#include "hip/hip_runtime.h"
// author: hao zhang, zhimin fan
#include "chrono_def.h"
#include "resultant_cuda.cuh"
#include <iostream>
#include <algorithm>
#include "polynomials.cuh"
#include "cyPolynomial.h"
#include "newton_cuda.cuh"

__constant__ double3 pD;
__constant__ double3 pL;

__constant__ int bezMat_deg;
__constant__ int bezMat_sz;

double get_bezout_matrix_time;
double eval_bezout_matrix_time;
double pick_section_time;
double bisection_solver_time;
double from_bezout_to_gamma_time;
double solve_back_time;

const int Cxz_deg = 10;

const int u2hat_deg = 5;
const int v2hat_deg = 5;
const int kappa2_deg = 4;

// #define BVP3_ALL_INIT() int i = blockIdx.x * blockDim.x + threadIdx.x;\
//                         BVP3<1> _pD = BVP3_init(pD);\
//                         BVP3<1> _pL = BVP3_init(pL);\
//                         BVP3<1> _p10 = BVP3_init(p10[i]);\
//                         BVP3<1> _n10 = BVP3_init(n10[i]);\
//                         BVP3<1> _p11 = BVP3_init(p11[i]);\
//                         BVP3<1> _n11 = BVP3_init(n11[i]);\
//                         BVP3<1> _p12 = BVP3_init(p12[i]);\
//                         BVP3<1> _n12 = BVP3_init(n12[i]);\
//                         BVP<2> u1, v1;\
//                         u1.a[0] = 0; u1.a[1] = 0; u1.a[2] = 1; u1.a[3] = 0;\
//                         v1.a[0] = 0; v1.a[1] = 1; v1.a[2] = 0; v1.a[3] = 0;\
//                         BVP3<2> t01 = BVP3_mul<1, 2, 2>(_p11, u1), t02 = BVP3_mul<1, 2, 2>(_p12, v1);\
//                         BVP3<2> t03 = BVP3_add<2, 2, 2>(t01, t02);\
//                         BVP3<2> x1 = BVP3_add<1, 2, 2>(_p10, t03);\
//                         BVP3<2> t04 = BVP3_mul<1, 2, 2>(_n11, u1), t05 = BVP3_mul<1, 2, 2>(_n12, v1);\
//                         BVP3<2> t06 = BVP3_add<2, 2, 2>(t04, t05);\
//                         BVP3<2> n1hat = BVP3_add<1, 2, 2>(_n10, t06);

// #define GET_BEZOUT(Dzy, Dxz)    divideByMax(Czy_); divideByMax(Cxz_); BVP<Cxz_deg> Czy = resize<Dzy, Cxz_deg>(Czy_); BVP<Cxz_deg> Cxz = resize<Dxz, Cxz_deg>(Cxz_);\
//                                 bezoutMatrix(Cxz, Czy, bezMats + i * (N_BEZOUT_TOTAL));\
//                                 for (int k = 0; k < Cxz_deg; k++)\
//                                     for (int j = 0; j < Cxz_deg; j++)\
//                                         Cxzs[i * MAXN * MAXN + k * Cxz_deg + j] = Cxz.a[k * Cxz_deg + j];

// __global__ void solve_one_1R(double3* __restrict__ p10, double3* __restrict__ n10,
//                             double3* __restrict__ p11, double3* __restrict__ n11,
//                             double3* __restrict__ p12, double3* __restrict__ n12,
//                             double* __restrict__ bezMats, double* __restrict__ Cxzs)
// {
//     BVP3_ALL_INIT()

//     BVP3<2> d0 = BVP3_sub<2, 1, 2>(x1, _pD);
//     BVP3<2> d1 = BVP3_sub<1, 2, 2>(_pL, x1);

//     // // 5 degree form
//     // // BVP d0_norm2 = BVP3_dot(d0, d0);
//     // // BVP d1_norm2 = BVP3_dot(d1, d1);
//     // // BVP3 c0 = BVP3_cross(d0, n1hat);
//     // // BVP3 c1 = BVP3_cross(d1, n1hat);
//     // // BVP3 c = BVP3_sub(BVP3_mul(BVP3_mul(c0, c0), d1_norm2), BVP3_mul(BVP3_mul(c1, c1), d0_norm2));

//     // // 4 degree form
//     BVP<3> d0_dot_n1hat = BVP3_dot<2, 2, 3>(d0, n1hat);
//     BVP<3> d1_dot_n1hat = BVP3_dot<2, 2, 3>(d1, n1hat);
//     BVP3<2> t1hat1 = BVP3_cross<2, 1, 2>(n1hat, _p11);
//     BVP3<2> t1hat2 = BVP3_cross<2, 1, 2>(n1hat, _p12);
//     BVP<3> d0_dot_t1hat2 = BVP3_dot<2, 2, 3>(d0, t1hat2);
//     BVP<3> d1_dot_t1hat2 = BVP3_dot<2, 2, 3>(d1, t1hat2);

//     BVP<5> Czy_; BVP<5> t1, t2;
//     bpm(d0_dot_n1hat, d1_dot_t1hat2, t1);
//     bpm(d0_dot_t1hat2, d1_dot_n1hat, t2);
//     bpa(t1, t2, Czy_);

//     BVP3<1> s = BVP3_sub<1, 1, 1>(_pL, _pD);
//     BVP3<3> cop = BVP3_cross<2, 2, 3>(BVP3_cross<2, 1, 2>(d0, s), BVP3_cross<2, 1, 2>(n1hat, s));
//     BVP<3> Cxz_, t3; bpa(cop.bvp[0], cop.bvp[1], t3); bpa(t3, cop.bvp[2], Cxz_);

//     GET_BEZOUT(5, 3)
// }

// __global__ void solve_one_2T(double3* p10, double3* n10,
//     double3* p11, double3* n11,
//     double3* p12, double3* n12,
//     double* bezMats, double* Cxzs)
// {
//     BVP3_ALL_INIT()
//     double eta = 1.5041833;

//     BVP3<2> d0 = BVP3_sub<2, 1, 2>(x1, _pD);
//     BVP3<2> d1 = BVP3_sub<1, 2, 2>(_pL, x1);
//     BVP<3> d0_norm2 = BVP3_dot<2, 2, 3>(d0, d0);
//     BVP<3> d1_norm2 = BVP3_dot<2, 2, 3>(d1, d1);
//     BVP3<3> c0 = BVP3_cross<2, 2, 3>(d0, n1hat);
//     BVP3<3> c1 = BVP3_cross<2, 2, 3>(d1, n1hat);
//     BVP<1> eeta; eeta.a[0] = eta * eta;
//     BVP3<7> c = BVP3_sub<7, 7, 7>(BVP3_mul<7, 1, 7>(BVP3_mul<5, 3, 7>(BVP3_mul<3, 3, 5>(c0, c0), d1_norm2), eeta), BVP3_mul<5, 3, 7>(BVP3_mul<3, 3, 5>(c1, c1), d0_norm2));
//     BVP<7> Czy_ = c.bvp[0];

//     BVP3<1> s = BVP3_sub<1, 1, 1>(_pL, _pD);
//     BVP3<3> cop = BVP3_cross<2, 2, 3>(BVP3_cross<2, 1, 2>(d0, s), BVP3_cross<2, 1, 2>(n1hat, s));
//     BVP<3> t1;
//     BVP<3> Cxz_; bpa(cop.bvp[0], cop.bvp[1], t1); bpa(t1, cop.bvp[2], Cxz_);

//     GET_BEZOUT(7, 3)
// }

__device__ void bezout_matrix(double *a, double *b, double *f)
{
    const int n = N_BVP - 1;
    for (size_t i = 0; i < n; ++i)
    {
        for (size_t j = i; j < n; ++j)
        {
            // a[i] * b[j+1] - a[j+1] * b[i]
            int addrai = (2 * N_BVP - i + 1) * (i) / 2;
            int addrbj1 = (2 * N_BVP2 - j) * (j + 1) / 2;
            int addraj1 = (2 * N_BVP - j) * (j + 1) / 2;
            int addrbi = (2 * N_BVP2 - i + 1) * (i) / 2;
            upm(a + addrai, b + addrbj1, f + bezout_addr(i, j), (N_BVP - i), (N_BVP2 - j - 1));
            upms(a + addraj1, b + addrbi, f + bezout_addr(i, j), (N_BVP - j - 1), (N_BVP2 - i));
        }
    }

    for (size_t i = 1; i < n - 1; ++i)
        for (size_t j = i; j < n - 1; ++j)
            upa(f + bezout_addr(i, j), f + bezout_addr(i - 1, j + 1), f + bezout_addr(i, j), N_BEZOUT_ITEM - i - j);
}

__global__ void solve(double3 xD, double3 xL,
                      double3 *p10, double3 *n10,
                      double3 *p11, double3 *n11,
                      double3 *p12, double3 *n12,
                      double3 *p20, double3 *n20,
                      double3 *p21, double3 *n21,
                      double3 *p22, double3 *n22,
                      double *bvpa, double *bvpb, double *bezout)
{
}

__global__ void solve_one_11RR(double3 *p10, double3 *n10,
                               double3 *p11, double3 *n11,
                               double3 *p12, double3 *n12,
                               double3 *p20, double3 *n20,
                               double3 *p21, double3 *n21,
                               double3 *p22, double3 *n22,
                               double *bezMats, double *Cxzs,
                               double *u2hats, double *v2hats, double *kappa2s)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double const1[1] = {1};
    double u1[3] = {0, 0, 1};
    double v1[3] = {0, 1, 0};
    double xD_x[1] = {pD.x};
    double xD_y[1] = {pD.y};
    double xD_z[1] = {pD.z};
    double xL_x[1] = {pL.x};
    double xL_y[1] = {pL.y};
    double xL_z[1] = {pL.z};
    double p10_x[1] = {p10[idx].x};
    double p10_y[1] = {p10[idx].y};
    double p10_z[1] = {p10[idx].z};
    double p11_x[1] = {p11[idx].x};
    double p11_y[1] = {p11[idx].y};
    double p11_z[1] = {p11[idx].z};
    double p12_x[1] = {p12[idx].x};
    double p12_y[1] = {p12[idx].y};
    double p12_z[1] = {p12[idx].z};
    double n10_x[1] = {n10[idx].x};
    double n10_y[1] = {n10[idx].y};
    double n10_z[1] = {n10[idx].z};
    double n11_x[1] = {n11[idx].x};
    double n11_y[1] = {n11[idx].y};
    double n11_z[1] = {n11[idx].z};
    double n12_x[1] = {n12[idx].x};
    double n12_y[1] = {n12[idx].y};
    double n12_z[1] = {n12[idx].z};
    double p20_x[1] = {p20[idx].x};
    double p20_y[1] = {p20[idx].y};
    double p20_z[1] = {p20[idx].z};
    double p21_x[1] = {p21[idx].x};
    double p21_y[1] = {p21[idx].y};
    double p21_z[1] = {p21[idx].z};
    double p22_x[1] = {p22[idx].x};
    double p22_y[1] = {p22[idx].y};
    double p22_z[1] = {p22[idx].z};
    double n20_x[1] = {n20[idx].x};
    double n20_y[1] = {n20[idx].y};
    double n20_z[1] = {n20[idx].z};
    double n21_x[1] = {n21[idx].x};
    double n21_y[1] = {n21[idx].y};
    double n21_z[1] = {n21[idx].z};
    double n22_x[1] = {n22[idx].x};
    double n22_y[1] = {n22[idx].y};
    double n22_z[1] = {n22[idx].z};
    double t1_x[3] = {0};
    bpm(u1, p11_x, t1_x, 2, 1, 2);
    double t1_y[3] = {0};
    bpm(u1, p11_y, t1_y, 2, 1, 2);
    double t1_z[3] = {0};
    bpm(u1, p11_z, t1_z, 2, 1, 2);
    bpma(v1, p12_x, t1_x, 1, 2, 1, 2);
    bpma(v1, p12_y, t1_y, 1, 2, 1, 2);
    bpma(v1, p12_z, t1_z, 1, 2, 1, 2);
    bpma(const1, p10_x, t1_x, 1, 1, 1, 2);
    bpma(const1, p10_y, t1_y, 1, 1, 1, 2);
    bpma(const1, p10_z, t1_z, 1, 1, 1, 2);
    double t2_x[3] = {0};
    bpm(u1, n11_x, t2_x, 2, 1, 2);
    double t2_y[3] = {0};
    bpm(u1, n11_y, t2_y, 2, 1, 2);
    double t2_z[3] = {0};
    bpm(u1, n11_z, t2_z, 2, 1, 2);
    bpma(v1, n12_x, t2_x, 1, 2, 1, 2);
    bpma(v1, n12_y, t2_y, 1, 2, 1, 2);
    bpma(v1, n12_z, t2_z, 1, 2, 1, 2);
    bpma(const1, n10_x, t2_x, 1, 1, 1, 2);
    bpma(const1, n10_y, t2_y, 1, 1, 1, 2);
    bpma(const1, n10_z, t2_z, 1, 1, 1, 2);
    double t4[1] = {-1.000000};
    double t3_x[1] = {0};
    bpm(t4, xD_x, t3_x, 1, 1, 1);
    double t3_y[1] = {0};
    bpm(t4, xD_y, t3_y, 1, 1, 1);
    double t3_z[1] = {0};
    bpm(t4, xD_z, t3_z, 1, 1, 1);
    double t5_x[3] = {0};
    bpa(t1_x, t3_x, t5_x, 2, 1, 2);
    double t5_y[3] = {0};
    bpa(t1_y, t3_y, t5_y, 2, 1, 2);
    double t5_z[3] = {0};
    bpa(t1_z, t3_z, t5_z, 2, 1, 2);
    double t6[6] = {0};
    bpm(t2_x, t2_x, t6, 2, 2, 3);
    bpma(t2_y, t2_y, t6, 1, 2, 2, 3);
    bpma(t2_z, t2_z, t6, 1, 2, 2, 3);
    double t7_x[10] = {0};
    bpm(t6, t5_x, t7_x, 3, 2, 4);
    double t7_y[10] = {0};
    bpm(t6, t5_y, t7_y, 3, 2, 4);
    double t7_z[10] = {0};
    bpm(t6, t5_z, t7_z, 3, 2, 4);
    double t8[6] = {0};
    bpm(t2_x, t5_x, t8, 2, 2, 3);
    bpma(t2_y, t5_y, t8, 1, 2, 2, 3);
    bpma(t2_z, t5_z, t8, 1, 2, 2, 3);
    bpma(t8, t2_x, t7_x, -2, 3, 2, 4);
    bpma(t8, t2_y, t7_y, -2, 3, 2, 4);
    bpma(t8, t2_z, t7_z, -2, 3, 2, 4);
    double t10[1] = {-1.000000};
    double t9_x[1] = {0};
    bpm(t10, p20_x, t9_x, 1, 1, 1);
    double t9_y[1] = {0};
    bpm(t10, p20_y, t9_y, 1, 1, 1);
    double t9_z[1] = {0};
    bpm(t10, p20_z, t9_z, 1, 1, 1);
    double t11_x[3] = {0};
    bpa(t1_x, t9_x, t11_x, 2, 1, 2);
    double t11_y[3] = {0};
    bpa(t1_y, t9_y, t11_y, 2, 1, 2);
    double t11_z[3] = {0};
    bpa(t1_z, t9_z, t11_z, 2, 1, 2);
    double t12_x[10] = {0};
    bpm(t7_y, p22_z, t12_x, 4, 1, 4);
    bpma(t7_z, p22_y, t12_x, -1, 4, 1, 4);
    double t12_y[10] = {0};
    bpm(t7_z, p22_x, t12_y, 4, 1, 4);
    bpma(t7_x, p22_z, t12_y, -1, 4, 1, 4);
    double t12_z[10] = {0};
    bpm(t7_x, p22_y, t12_z, 4, 1, 4);
    bpma(t7_y, p22_x, t12_z, -1, 4, 1, 4);
    double t13[15] = {0};
    bpm(t12_x, t11_x, t13, 4, 2, 5);
    bpma(t12_y, t11_y, t13, 1, 4, 2, 5);
    bpma(t12_z, t11_z, t13, 1, 4, 2, 5);
    double t14_x[3] = {0};
    bpm(t11_y, p21_z, t14_x, 2, 1, 2);
    bpma(t11_z, p21_y, t14_x, -1, 2, 1, 2);
    double t14_y[3] = {0};
    bpm(t11_z, p21_x, t14_y, 2, 1, 2);
    bpma(t11_x, p21_z, t14_y, -1, 2, 1, 2);
    double t14_z[3] = {0};
    bpm(t11_x, p21_y, t14_z, 2, 1, 2);
    bpma(t11_y, p21_x, t14_z, -1, 2, 1, 2);
    double t15[15] = {0};
    bpm(t14_x, t7_x, t15, 2, 4, 5);
    bpma(t14_y, t7_y, t15, 1, 2, 4, 5);
    bpma(t14_z, t7_z, t15, 1, 2, 4, 5);
    double t16_x[10] = {0};
    bpm(t7_y, p22_z, t16_x, 4, 1, 4);
    bpma(t7_z, p22_y, t16_x, -1, 4, 1, 4);
    double t16_y[10] = {0};
    bpm(t7_z, p22_x, t16_y, 4, 1, 4);
    bpma(t7_x, p22_z, t16_y, -1, 4, 1, 4);
    double t16_z[10] = {0};
    bpm(t7_x, p22_y, t16_z, 4, 1, 4);
    bpma(t7_y, p22_x, t16_z, -1, 4, 1, 4);
    double t17[10] = {0};
    bpm(t16_x, p21_x, t17, 4, 1, 4);
    bpma(t16_y, p21_y, t17, 1, 4, 1, 4);
    bpma(t16_z, p21_z, t17, 1, 4, 1, 4);
    double t18_x[15] = {0};
    bpm(t13, p21_x, t18_x, 5, 1, 5);
    double t18_y[15] = {0};
    bpm(t13, p21_y, t18_y, 5, 1, 5);
    double t18_z[15] = {0};
    bpm(t13, p21_z, t18_z, 5, 1, 5);
    bpma(t15, p22_x, t18_x, 1, 5, 1, 5);
    bpma(t15, p22_y, t18_y, 1, 5, 1, 5);
    bpma(t15, p22_z, t18_z, 1, 5, 1, 5);
    bpma(t17, p20_x, t18_x, 1, 4, 1, 5);
    bpma(t17, p20_y, t18_y, 1, 4, 1, 5);
    bpma(t17, p20_z, t18_z, 1, 4, 1, 5);
    double t20[1] = {-1.000000};
    double t19_x[15] = {0};
    bpm(t20, t18_x, t19_x, 1, 5, 5);
    double t19_y[15] = {0};
    bpm(t20, t18_y, t19_y, 1, 5, 5);
    double t19_z[15] = {0};
    bpm(t20, t18_z, t19_z, 1, 5, 5);
    bpma(t17, xL_x, t19_x, 1, 4, 1, 5);
    bpma(t17, xL_y, t19_y, 1, 4, 1, 5);
    bpma(t17, xL_z, t19_z, 1, 4, 1, 5);
    double t21_x[15] = {0};
    bpm(t13, n21_x, t21_x, 5, 1, 5);
    double t21_y[15] = {0};
    bpm(t13, n21_y, t21_y, 5, 1, 5);
    double t21_z[15] = {0};
    bpm(t13, n21_z, t21_z, 5, 1, 5);
    bpma(t15, n22_x, t21_x, 1, 5, 1, 5);
    bpma(t15, n22_y, t21_y, 1, 5, 1, 5);
    bpma(t15, n22_z, t21_z, 1, 5, 1, 5);
    bpma(t17, n20_x, t21_x, 1, 4, 1, 5);
    bpma(t17, n20_y, t21_y, 1, 4, 1, 5);
    bpma(t17, n20_z, t21_z, 1, 4, 1, 5);
    double t22_x[15] = {0};
    bpm(t21_y, p21_z, t22_x, 5, 1, 5);
    bpma(t21_z, p21_y, t22_x, -1, 5, 1, 5);
    double t22_y[15] = {0};
    bpm(t21_z, p21_x, t22_y, 5, 1, 5);
    bpma(t21_x, p21_z, t22_y, -1, 5, 1, 5);
    double t22_z[15] = {0};
    bpm(t21_x, p21_y, t22_z, 5, 1, 5);
    bpma(t21_y, p21_x, t22_z, -1, 5, 1, 5);
    double t23_x[15] = {0};
    bpm(t21_y, p22_z, t23_x, 5, 1, 5);
    bpma(t21_z, p22_y, t23_x, -1, 5, 1, 5);
    double t23_y[15] = {0};
    bpm(t21_z, p22_x, t23_y, 5, 1, 5);
    bpma(t21_x, p22_z, t23_y, -1, 5, 1, 5);
    double t23_z[15] = {0};
    bpm(t21_x, p22_y, t23_z, 5, 1, 5);
    bpma(t21_y, p22_x, t23_z, -1, 5, 1, 5);
    double t24[36] = {0};
    bpm(t7_x, t21_x, t24, 4, 5, 8);
    bpma(t7_y, t21_y, t24, 1, 4, 5, 8);
    bpma(t7_z, t21_z, t24, 1, 4, 5, 8);
    double t25[45] = {0};
    bpm(t19_x, t21_x, t25, 5, 5, 9);
    bpma(t19_y, t21_y, t25, 1, 5, 5, 9);
    bpma(t19_z, t21_z, t25, 1, 5, 5, 9);
    double t26[45] = {0};
    bpm(t19_x, t23_x, t26, 5, 5, 9);
    bpma(t19_y, t23_y, t26, 1, 5, 5, 9);
    bpma(t19_z, t23_z, t26, 1, 5, 5, 9);
    double t27[136] = {0};
    bpm(t24, t26, t27, 8, 9, 16);
    double t28[36] = {0};
    bpm(t7_x, t23_x, t28, 4, 5, 8);
    bpma(t7_y, t23_y, t28, 1, 4, 5, 8);
    bpma(t7_z, t23_z, t28, 1, 4, 5, 8);
    bpma(t25, t28, t27, 1, 9, 8, 16);
    double t30[1] = {-1.000000};
    double t29_x[3] = {0};
    bpm(t30, t1_x, t29_x, 1, 2, 2);
    double t29_y[3] = {0};
    bpm(t30, t1_y, t29_y, 1, 2, 2);
    double t29_z[3] = {0};
    bpm(t30, t1_z, t29_z, 1, 2, 2);
    double t31_x[3] = {0};
    bpa(t29_x, xL_x, t31_x, 2, 1, 2);
    double t31_y[3] = {0};
    bpa(t29_y, xL_y, t31_y, 2, 1, 2);
    double t31_z[3] = {0};
    bpa(t29_z, xL_z, t31_z, 2, 1, 2);
    double t32_x[21] = {0};
    bpm(t19_y, t31_z, t32_x, 5, 2, 6);
    bpma(t19_z, t31_y, t32_x, -1, 5, 2, 6);
    double t32_y[21] = {0};
    bpm(t19_z, t31_x, t32_y, 5, 2, 6);
    bpma(t19_x, t31_z, t32_y, -1, 5, 2, 6);
    double t32_z[21] = {0};
    bpm(t19_x, t31_y, t32_z, 5, 2, 6);
    bpma(t19_y, t31_x, t32_z, -1, 5, 2, 6);
    double t33[55] = {0};
    bpm(t32_x, t21_x, t33, 6, 5, 10);
    bpma(t32_y, t21_y, t33, 1, 6, 5, 10);
    bpma(t32_z, t21_z, t33, 1, 6, 5, 10);
    // for (int i = 0; i < 136; i++)
    //     bvpa[idx * N_BVPA + i] = t27[i];
    double sum = 0;
    const double normal = 1e-16; // make this larger leads to slower but more accurate solve_cuda kernels
    for (int i = 0; i < N_BVPA; i++)
        sum += abs(t27[i]);
    sum /= normal;
    for (int i = 0; i < N_BVPA; i++)
        t27[i] /= sum;
    sum = 0;
    for (int i = 0; i < N_BVPB; i++)
        sum += abs(t33[i]);
    sum /= normal;
    for (int i = 0; i < N_BVPB; i++)
        t33[i] /= sum;

    for (int i = 0; i < N_BVPB; i++)
        Cxzs[idx * N_BVPB + i] = t33[i];
    for (int i = 0; i < N_BVPU; i++)
        u2hats[idx * N_BVPU + i] = t13[i];
    for (int i = 0; i < N_BVPU; i++)
        v2hats[idx * N_BVPU + i] = t15[i];
    for (int i = 0; i < N_BVPK; i++)
        kappa2s[idx * N_BVPK + i] = t17[i];
    bezout_matrix(t27, t33, bezMats + idx * N_BEZOUT_TOTAL);
    // ! bug: Cxzs[0][0] not consistent with CPU version
    // double* bezMats, double* Cxzs,
    // double* u2hats, double* v2hats, double* kappa2s)
}

__global__ void eval_bezout_matrix(hipTextureObject_t bezMatsTex, double *__restrict__ values)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    int i = ind / (global_poly_cutoff + 1), j = ind % (global_poly_cutoff + 1);
    double pos = j * 1.0 / global_poly_cutoff;
    values[ind] = evalMatrixPolynomialDeterminantTex(bezMatsTex, bezMat_sz, bezMat_deg, i, pos);
}

__global__ void bisection_solver(hipTextureObject_t bezMatsTex, int *__restrict__ segments, double *__restrict__ f, double *__restrict__ res)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    int x = segments[ind];
    int i = x / (global_poly_cutoff + 1), j = x % (global_poly_cutoff + 1);
    double l = j * 1.0 / global_poly_cutoff, r = (j + 1) * 1.0 / global_poly_cutoff;
    double mid = (l + r) / 2;
    for (int _ = 1; _ < max_iteration_dichotomy_cuda; _++)
    {
        double fm = evalMatrixPolynomialDeterminantTex(bezMatsTex, bezMat_sz, bezMat_deg, i, mid);
        if (fm * f[ind] > 0)
            r = mid;
        else
            l = mid;
        mid = (l + r) / 2;
    }
    res[ind] = mid;
}

int get_bvp_idx(int i, int j, int n)
{
    return i * (n + n - i + 1) / 2 + j;
}

std::vector<std::tuple<double, double, double, double>> solve_back_1(std::vector<double> &result, double *Cxzs, int Cxz_deg, double global_poly_cutoff_eps)
{
    std::vector<std::tuple<double, double, double, double>> sols;
    double roots2[9 * (N_POLY - 1)];
    double doubleervalMin = 0;
    double doubleervalMax = 1;
    double errorThreshold = 1e-9;
    double x_tolerance = 0;
    for (double gamma : result)
    {
        double uniploy[N_DEGREE] = {0};
        for (int i = 0; i < Cxz_deg; i++)
        {
            uniploy[i] = 0;
            for (int j = Cxz_deg - 1 - i; j >= 0; j--)
                uniploy[i] = uniploy[i] * gamma + Cxzs[get_bvp_idx(i, j, Cxz_deg)];
        }

        double prefix_max[N_DEGREE] = {0}, suffix_max[N_DEGREE];
        for (int i = 0; i < N_DEGREE; i++)
            prefix_max[i] = suffix_max[i] = uniploy[i];
        for (int i = 1; i < N_DEGREE; i++)
            prefix_max[i] = max(prefix_max[i], prefix_max[i - 1]);
        for (int i = N_DEGREE - 1; i >= 0; i--)
            suffix_max[i] = max(suffix_max[i], suffix_max[i + 1]);

        int truncate_degree = 9;
        for (int i = 3; i < N_DEGREE - 1; i++)
        {
            if (suffix_max[i + 1] * 1e-9 < prefix_max[i])
            {
                truncate_degree = i;
                break;
            }
        }

        int numRoot2 = 0;
        if (truncate_degree <= 4)
        {
            const int const_deg = 4;
            cy::Polynomial<double, const_deg> poly2;
            for (int k = 0; k <= const_deg; k++)
                poly2.coef[k] = uniploy[k];
            numRoot2 = poly2.Roots(roots2, doubleervalMin, doubleervalMax, errorThreshold);
        }
        else if (truncate_degree <= 5)
        {
            const int const_deg = 5;
            cy::Polynomial<double, const_deg> poly2;
            for (int k = 0; k <= const_deg; k++)
                poly2.coef[k] = uniploy[k];
            numRoot2 = poly2.Roots(roots2, doubleervalMin, doubleervalMax, errorThreshold);
        }
        else if (truncate_degree <= 6)
        {
            const int const_deg = 6;
            cy::Polynomial<double, const_deg> poly2;
            for (int k = 0; k <= const_deg; k++)
                poly2.coef[k] = uniploy[k];
            numRoot2 = poly2.Roots(roots2, doubleervalMin, doubleervalMax, errorThreshold);
        }
        else if (truncate_degree <= 7)
        {
            const int const_deg = 7;
            cy::Polynomial<double, const_deg> poly2;
            for (int k = 0; k <= const_deg; k++)
                poly2.coef[k] = uniploy[k];
            numRoot2 = poly2.Roots(roots2, doubleervalMin, doubleervalMax, errorThreshold);
        }
        else if (truncate_degree <= 8)
        {
            const int const_deg = 8;
            cy::Polynomial<double, const_deg> poly2;
            for (int k = 0; k <= const_deg; k++)
                poly2.coef[k] = uniploy[k];
            numRoot2 = poly2.Roots(roots2, doubleervalMin, doubleervalMax, errorThreshold);
        }
        else
        {
            const int const_deg = 9;
            cy::Polynomial<double, const_deg> poly2;
            for (int k = 0; k <= const_deg; k++)
                poly2.coef[k] = uniploy[k];
            numRoot2 = poly2.Roots(roots2, doubleervalMin, doubleervalMax, errorThreshold);
        }

        for (int i = 0; i < numRoot2; i++)
        {
            double beta = roots2[i];
            if (beta < -x_tolerance || beta > 1 + x_tolerance || beta + gamma > 1 + x_tolerance)
                continue;
            double alpha = 1.0 - beta - gamma;
            sols.push_back(std::make_tuple(alpha, beta, 0.333, 0.333));
        }
    }
    return sols;
}

std::vector<std::tuple<double, double, double, double>> solve_back_2(std::vector<double> &result, double *Cxzs, int Cxz_deg, double *u2hat, double *v2hat, double *kappa2, double global_poly_cutoff_eps)
{
    std::vector<std::tuple<double, double, double, double>> sols = solve_back_1(result, Cxzs, Cxz_deg, global_poly_cutoff_eps), sols2;
    int sols_cnt = sols.size();
    for (int i = 0; i < sols_cnt; i++)
    {
        double uniploy[N_DEGREE] = {0};
        double alpha = std::get<0>(sols[i]), beta = std::get<1>(sols[i]);
        double gamma = 1.0 - alpha - beta;
        for (int j = 0; j < u2hat_deg; j++)
        {
            uniploy[j] = 0;
            for (int k = u2hat_deg - 1 - j; k >= 0; k--)
                uniploy[j] = uniploy[j] * gamma + u2hat[get_bvp_idx(j, k, u2hat_deg)];
        }
        double u2hat_val = 0.0;
        for (int j = u2hat_deg - 1; j >= 0; j--)
            u2hat_val = u2hat_val * beta + uniploy[j];

        for (int j = 0; j < v2hat_deg; j++)
        {
            uniploy[j] = 0;
            for (int k = v2hat_deg - 1 - j; k >= 0; k--)
                uniploy[j] = uniploy[j] * gamma + v2hat[get_bvp_idx(j, k, v2hat_deg)];
        }
        double v2hat_val = 0.0;
        for (int j = v2hat_deg - 1; j >= 0; j--)
            v2hat_val = v2hat_val * beta + uniploy[j];

        for (int j = 0; j < kappa2_deg; j++)
        {
            uniploy[j] = 0;
            for (int k = kappa2_deg - 1 - j; k >= 0; k--)
                uniploy[j] = uniploy[j] * gamma + kappa2[get_bvp_idx(j, k, kappa2_deg)];
        }
        double kappa2_val = 0.0;
        for (int j = kappa2_deg - 1; j >= 0; j--)
            kappa2_val = kappa2_val * beta + uniploy[j];

        double beta2 = u2hat_val / kappa2_val;
        double gamma2 = v2hat_val / kappa2_val;

        double alpha2 = 1 - beta2 - gamma2;

        if (alpha <= 0 || beta <= 0 || gamma <= 0)
            continue;
        if (alpha2 <= 0 || beta2 <= 0 || gamma2 <= 0)
            continue;
        sols2.push_back(std::make_tuple(alpha, beta, alpha2, beta2));
    }
    return sols2;
}

std::vector<std::vector<std::tuple<double, double, double, double>>> solve_cuda(
    int chain_type, int N,
    double *bezMat_, int bezMat_deg_, int bezMat_sz_,
    double *Cxzs_, int Cxz_deg_,
    double *u2hat_, double *v2hat_, double *kappa2_,
    int cutoff_resultant, float cutoff_eps_resultant, int maxIterationDichotomy)
{
    double *bezMats;
    hipMalloc((void **)&bezMats, N * (N_BEZOUT_TOTAL) * sizeof(double));
    hipMemcpy(bezMats, bezMat_, N * (N_BEZOUT_TOTAL) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(global_poly_cutoff), &cutoff_resultant, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(bezMat_deg), &bezMat_deg_, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(bezMat_sz), &bezMat_sz_, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(max_iteration_dichotomy_cuda), &maxIterationDichotomy, sizeof(int));
    int threadsPerBlock = 32;

    hipResourceDesc bezMatsResDesc;
    memset(&bezMatsResDesc, 0, sizeof(bezMatsResDesc));
    bezMatsResDesc.resType = hipResourceTypeLinear;
    bezMatsResDesc.res.linear.devPtr = bezMats;
    bezMatsResDesc.res.linear.desc.f = hipChannelFormatKindSigned;
    bezMatsResDesc.res.linear.desc.x = 32;
    bezMatsResDesc.res.linear.desc.y = 32;
    bezMatsResDesc.res.linear.sizeInBytes = N * (N_BEZOUT_TOTAL) * sizeof(double);

    hipTextureDesc bezMatsTexDesc;
    memset(&bezMatsTexDesc, 0, sizeof(bezMatsTexDesc));
    bezMatsTexDesc.readMode = hipReadModeElementType;

    hipTextureObject_t bezMatsTex;
    hipCreateTextureObject(&bezMatsTex, &bezMatsResDesc, &bezMatsTexDesc, NULL);

    double *values;
    hipMalloc((void **)&values, N * (cutoff_resultant + 1) * sizeof(double));
    int blocksPerGrid = (N * (cutoff_resultant + 1) - 1) / threadsPerBlock + 1;
    auto start = CHRONO_NOW;
    auto start_ = start;
    eval_bezout_matrix<<<blocksPerGrid, threadsPerBlock>>>(bezMatsTex, values);
    // this kernel uses float precision for efficiency
    // it will be better to recheck the chosen intervals using double precision
    hipDeviceSynchronize();
    auto end = CHRONO_NOW;
    eval_bezout_matrix_time = CHRONO_DIFF(end - start) * 1e-3;

    double *v = new double[N * (cutoff_resultant + 1)];
    hipMemcpy(v, values, N * (cutoff_resultant + 1) * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(values);

    std::vector<int> segments_;
    std::vector<double> f_;
    std::vector<std::vector<double>> results;
    results.resize(N);

    start = CHRONO_NOW;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < cutoff_resultant; j++)
        {
            int ind = i * (cutoff_resultant + 1) + j;
            if (v[ind] == 0)
            {
                results[i].push_back(v[ind]);
                continue;
            }
            if (v[ind] * v[ind + 1] <= 0 && v[ind] != 0 && v[ind + 1] != 0)
            {
                segments_.push_back(ind);
                f_.push_back(v[ind] < 0 ? 1 : -1);
            }
        }
        if (v[i * (cutoff_resultant + 1) + cutoff_resultant] == 0)
        {
            results[i].push_back(v[i * (cutoff_resultant + 1) + cutoff_resultant]);
        }
    }
    end = CHRONO_NOW;
    pick_section_time = CHRONO_DIFF(end - start) * 1e-3;

    std::vector<std::vector<std::tuple<double, double, double, double>>> sols;
    sols.resize(N);
    if (segments_.empty())
    {
        hipFree(bezMats);
        hipDestroyTextureObject(bezMatsTex);
        return sols;
    }

    int *segments;
    double *res, *f;
    hipMalloc((void **)&segments, segments_.size() * sizeof(int));
    hipMalloc((void **)&res, segments_.size() * sizeof(double));
    hipMalloc((void **)&f, segments_.size() * sizeof(double));
    hipMemcpy(segments, segments_.data(), segments_.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(f, f_.data(), f_.size() * sizeof(double), hipMemcpyHostToDevice);
    blocksPerGrid = (segments_.size() - 1) / threadsPerBlock + 1;

    start = CHRONO_NOW;
    bisection_solver<<<blocksPerGrid, threadsPerBlock>>>(bezMatsTex, segments, f, res);
    hipDeviceSynchronize();
    end = CHRONO_NOW;
    bisection_solver_time = CHRONO_DIFF(end - start) * 1e-3;
    from_bezout_to_gamma_time = CHRONO_DIFF(end - start_) * 1e-3;

    double *res_ = new double[segments_.size()];
    hipMemcpy(res_, res, segments_.size() * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < segments_.size(); i++)
        results[segments_[i] / (cutoff_resultant + 1)].push_back(res_[i]);

    hipFree(bezMats);
    hipFree(segments);
    hipFree(res);
    hipDestroyTextureObject(bezMatsTex);

    start = CHRONO_NOW;
    int cnt = 0;
    for (std::vector<double> &v : results)
    {
        if (chain_type < 10)
            sols[cnt] = (solve_back_1(v, Cxzs_ + cnt * N_BVPB, Cxz_deg_, cutoff_eps_resultant));
        else
            sols[cnt] = (solve_back_2(v, Cxzs_ + cnt * N_BVPB, Cxz_deg_, u2hat_ + cnt * N_BVPU, v2hat_ + cnt * N_BVPU, kappa2_ + cnt * N_BVPK, cutoff_eps_resultant));
        cnt++;
    }
    end = CHRONO_NOW;
    solve_back_time = CHRONO_DIFF(end - start) * 1e-3;

    return sols;
}

std::vector<std::vector<std::tuple<double, double, double, double>>> solve_cuda(
    int chain_type, // 1R, 2T, 11RR, ...
    double3 &pD_,
    double3 &pL_,
    const std::vector<double3> &p10_,
    const std::vector<double3> &n10_,
    const std::vector<double3> &p11_,
    const std::vector<double3> &n11_,
    const std::vector<double3> &p12_,
    const std::vector<double3> &n12_,
    const std::vector<double3> &p20_,
    const std::vector<double3> &n20_,
    const std::vector<double3> &p21_,
    const std::vector<double3> &n21_,
    const std::vector<double3> &p22_,
    const std::vector<double3> &n22_,
    int cutoff_matrix,
    int cutoff_resultant,
    float cutoff_eps_resultant,
    int maxIterationDichotomy)
{
    int N = p10_.size();
    int threadsPerBlock = 64;
    int blocksPerGrid = (N - 1) / threadsPerBlock + 1;

    hipMemcpyToSymbol(HIP_SYMBOL(pD), &pD_, sizeof(double3));
    hipMemcpyToSymbol(HIP_SYMBOL(pL), &pL_, sizeof(double3));

    hipMemcpyToSymbol(HIP_SYMBOL(global_poly_cutoff), &cutoff_resultant, sizeof(int));
    cutoff_matrix++;
    if (N_MAT < cutoff_matrix)
        cutoff_matrix = N_MAT;
    // hipMemcpyToSymbol(HIP_SYMBOL(global_mat_cutoff), &cutoff_matrix, sizeof(int)); DO NOT SUPPORT
    hipMemcpyToSymbol(HIP_SYMBOL(max_iteration_dichotomy_cuda), &maxIterationDichotomy, sizeof(int));

    if (cutoff_matrix != global_mat_cutoff)
        std::cout << "Warning: parameter cutoff_matrix is not equal to the builtin global_mat_cutoff !!!" << std::endl;

    double3 *p10, *n10, *p11, *n11, *p12, *n12, *p20, *n20, *p21, *n21, *p22, *n22;
    hipMalloc((void **)&p10, N * sizeof(double3));
    hipMemcpy(p10, p10_.data(), N * sizeof(double3), hipMemcpyHostToDevice);
    hipMalloc((void **)&n10, N * sizeof(double3));
    hipMemcpy(n10, n10_.data(), N * sizeof(double3), hipMemcpyHostToDevice);
    hipMalloc((void **)&p11, N * sizeof(double3));
    hipMemcpy(p11, p11_.data(), N * sizeof(double3), hipMemcpyHostToDevice);
    hipMalloc((void **)&n11, N * sizeof(double3));
    hipMemcpy(n11, n11_.data(), N * sizeof(double3), hipMemcpyHostToDevice);
    hipMalloc((void **)&p12, N * sizeof(double3));
    hipMemcpy(p12, p12_.data(), N * sizeof(double3), hipMemcpyHostToDevice);
    hipMalloc((void **)&n12, N * sizeof(double3));
    hipMemcpy(n12, n12_.data(), N * sizeof(double3), hipMemcpyHostToDevice);

    double *bezMats;
    hipMalloc((void **)&bezMats, N * (N_BEZOUT_TOTAL) * sizeof(double));
    int bezMat_deg_ = 24, bezMat_sz_ = 15;
    hipMemcpyToSymbol(HIP_SYMBOL(bezMat_deg), &bezMat_deg_, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(bezMat_sz), &bezMat_sz_, sizeof(int));

    double *Cxzs;
    hipMalloc((void **)&Cxzs, N * N_BVPB * sizeof(double));

    double *u2hat, *v2hat, *kappa2;

    auto start = CHRONO_NOW;

    // if (chain_type == 1) {
    //     solve_one_1R<<<blocksPerGrid, threadsPerBlock>>>(p10, n10, p11, n11, p12, n12, bezMats, Cxzs);
    //     hipDeviceSynchronize();
    // }
    // if (chain_type == 2) {
    //     solve_one_2T<<<blocksPerGrid, threadsPerBlock>>>(p10, n10, p11, n11, p12, n12, bezMats, Cxzs);
    //     hipDeviceSynchronize();
    // }
    if (chain_type == 11)
    {
        hipMalloc((void **)&u2hat, N * N_BVPU * sizeof(double));
        hipMalloc((void **)&v2hat, N * N_BVPU * sizeof(double));
        hipMalloc((void **)&kappa2, N * N_BVPK * sizeof(double));
        hipMalloc((void **)&p20, N * sizeof(double3));
        hipMemcpy(p20, p20_.data(), N * sizeof(double3), hipMemcpyHostToDevice);
        hipMalloc((void **)&n20, N * sizeof(double3));
        hipMemcpy(n20, n20_.data(), N * sizeof(double3), hipMemcpyHostToDevice);
        hipMalloc((void **)&p21, N * sizeof(double3));
        hipMemcpy(p21, p21_.data(), N * sizeof(double3), hipMemcpyHostToDevice);
        hipMalloc((void **)&n21, N * sizeof(double3));
        hipMemcpy(n21, n21_.data(), N * sizeof(double3), hipMemcpyHostToDevice);
        hipMalloc((void **)&p22, N * sizeof(double3));
        hipMemcpy(p22, p22_.data(), N * sizeof(double3), hipMemcpyHostToDevice);
        hipMalloc((void **)&n22, N * sizeof(double3));
        hipMemcpy(n22, n22_.data(), N * sizeof(double3), hipMemcpyHostToDevice);

        solve_one_11RR<<<blocksPerGrid, threadsPerBlock>>>(p10, n10, p11, n11, p12, n12, p20, n20, p21, n21, p22, n22, bezMats, Cxzs, u2hat, v2hat, kappa2);
        hipDeviceSynchronize();
        hipFree(p20);
        hipFree(n20);
        hipFree(p21);
        hipFree(n21);
        hipFree(p22);
        hipFree(n22);
    }
    // TODO: 22TT
    auto end = CHRONO_NOW;
    get_bezout_matrix_time = CHRONO_DIFF(end - start) * 1e-3;

    hipFree(p10);
    hipFree(n10);
    hipFree(p11);
    hipFree(n11);
    hipFree(p12);
    hipFree(n12);

    double *u2hat_, *v2hat_, *kappa2_;

    if (chain_type > 10)
    {
        u2hat_ = new double[N * N_BVPU];
        v2hat_ = new double[N * N_BVPU];
        kappa2_ = new double[N * N_BVPK];
        hipMemcpy(u2hat_, u2hat, N * N_BVPU * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(v2hat_, v2hat, N * N_BVPU * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(kappa2_, kappa2, N * N_BVPK * sizeof(double), hipMemcpyDeviceToHost);
        hipFree(u2hat);
        hipFree(v2hat);
        hipFree(kappa2);
    }

    double *Cxzs_ = new double[N * N_BVPB];
    hipMemcpy(Cxzs_, Cxzs, N * N_BVPB * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(Cxzs);

    double *bezMats_ = new double[N * (N_BEZOUT_TOTAL)];
    hipMemcpy(bezMats_, bezMats, N * (N_BEZOUT_TOTAL) * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(bezMats);

    return solve_cuda(chain_type, N, bezMats_, bezMat_deg_, bezMat_sz_, Cxzs_, Cxz_deg, u2hat_, v2hat_, kappa2_, cutoff_resultant, cutoff_eps_resultant, maxIterationDichotomy);
}
